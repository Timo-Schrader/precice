#include "hip/hip_runtime.h"
#ifndef PRECICE_NO_GINKGO

#include <cmath>
#include <stdio.h>
// #include "math/math.hpp"

#include "mapping/impl/DeviceBasisFunctions.cuh"

#define NUMERICAL_ZERO_DIFFERENCE 1.0e-14

namespace math {
template <int iexp, typename T>
inline constexpr T pow_int(const T x)
{
  static_assert(iexp >= 0, "Exponent must be an integer greater or equal to zero.");

  if (iexp == 0)
    return static_cast<T>(1.);
  else
    // exponentiation by squaring
    return ((iexp % 2 == 1) ? x * pow_int<iexp / 2>(x * x) : pow_int<iexp / 2>(x * x));
}
} // namespace math

namespace precice {
namespace mapping {

SHARED_HOST_DEVICE_FUNCTION double ThinPlateSplinesFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  // We don't need to read any values from params since there is no need here
  return std::log(std::max(radius, NUMERICAL_ZERO_DIFFERENCE)) * math::pow_int<2>(radius);
}

SHARED_HOST_DEVICE_FUNCTION double MultiQuadraticsFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  double cPow2 = params.at(0);
  return std::sqrt(cPow2 + math::pow_int<2>(radius));
}

SHARED_HOST_DEVICE_FUNCTION double InverseMultiquadricsFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  double cPow2 = params.at(0);
  return 1.0 / std::sqrt(cPow2 + math::pow_int<2>(radius));
}

SHARED_HOST_DEVICE_FUNCTION double VolumeSplinesFunctor::operator()(const double radius, const std::array<double, 3> params) const
{
  return std::abs(radius);
}

SHARED_HOST_DEVICE_FUNCTION double GaussianFunctor::operator()(const double radius, std::array<double, 3> params) const
{
  double shape         = params.at(0);
  double supportRadius = params.at(1);
  double deltaY        = params.at(2);

  if (radius > supportRadius) {
    return 0.0;
  } else {
    return std::exp(-math::pow_int<2>(shape * radius)) - deltaY;
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactThinPlateSplinesC2Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return 1.0 - 30.0 * std::pow(p, 2) - 10.0 * std::pow(p, 3) + 45.0 * std::pow(p, 4) - 6.0 * std::pow(p, 5) - std::pow(p, 3) * 60.0 * std::log(std::max(p, NUMERICAL_ZERO_DIFFERENCE));
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC0Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 2);
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC2Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 4) * (4 * p + 1);
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC4Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    return std::pow(1.0 - p, 6) * (35 * std::pow(p, 2) + 18 * p + 3, 2);
  }
}

SHARED_HOST_DEVICE_FUNCTION double CompactPolynomialC6Functor::operator()(const double radius, const std::array<double, 3> params) const
{
  double       r_inv = params.at(0);
  double const p     = radius * r_inv;
  if (p >= 1) {
    return 0.0;
  } else {
    double result = fma(8.0, p, 1.0);
    result        = fma(25.0, math::pow_int<2>(p), result);
    result        = fma(32.0, math::pow_int<3>(p), result);
    return result * math::pow_int<8>(1.0 - p);
  }
}

} // namespace mapping
} // namespace precice

#endif
