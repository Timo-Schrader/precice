#include "hip/hip_runtime.h"
#include "ginkgo/ginkgo.hpp"
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "utils/Event.hpp"
#include "utils/EventUtils.hpp"

using GinkgoMatrix = gko::matrix::Dense<>;

// Handles for low-level CUDA libraries
hipsolverHandle_t solverHandle;
hipblasHandle_t cublasHandle;
hipblasStatus_t hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolverStatus = HIPSOLVER_STATUS_SUCCESS;
hipError_t cudaErrorCode = hipSuccess;

// Important variables which track the state of the solver routines
double *dTau = nullptr;
double *dWork = nullptr;
int *devInfo = nullptr;

void initCuSolver(const bool enableUnifiedMemory=false){
    // Allocating important CUDA variables
    if(enableUnifiedMemory){
        hipMallocManaged((void **)&dWork, sizeof(double));
        hipMallocManaged((void **)&devInfo, sizeof(int));
        hipMallocManaged((void **)&dTau, sizeof(double));
    }
    else {
        hipMalloc((void **)&dWork, sizeof(double));
        hipMalloc((void **)&devInfo, sizeof(int));
        hipMalloc((void **)&dTau, sizeof(double));
    }

}

void deInitCuSolver(){
    // Freeing CUDA variables
    hipFree(dTau);
    hipFree(dWork);
    hipFree(devInfo);
}

void computeQR(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *const A, GinkgoMatrix *Q, GinkgoMatrix *R)
{
    hipsolverDnCreate(&solverHandle);
    hipblasCreate(&cublasHandle);

    // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
    // Making a copy since every value will be overridden
    auto A_T = gko::share(GinkgoMatrix::create(exec, gko::dim<2>(A->get_size()[1], A->get_size()[0])));
    A->transpose(gko::lend(A_T));

    // Setting dimensions for solver
    const unsigned int M = A_T->get_size()[1];
    const unsigned int N = A_T->get_size()[0];

    auto temp_A = gko::share(GinkgoMatrix::create(exec, gko::dim<2>{M, N}));

    const int lda = max(1, M);
    const int k = min(M, N);

    int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;

    precice::utils::Event calculateQRDecompEvent{"calculateQRDecomp"};

    // Query working space of geqrf and orgqr
    cusolverStatus = hipsolverDnDgeqrf_bufferSize(solverHandle, M, N, A_T->get_values(), lda, &lwork_geqrf);
    assert(cusolverStatus == cusolverStatus_SUCCESS);
    cusolverStatus = hipsolverDnDorgqr_bufferSize(solverHandle, M, N, k, A_T->get_values(), lda, dTau, &lwork_orgqr);
    assert(cusolverStatus == cusolverStatus_SUCCESS);
    lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
    cudaErrorCode = hipMalloc((void **)&dWork, sizeof(double) * lwork);
    assert(hipSuccess == cudaErrorCode);

    // Compute QR factorization
    cusolverStatus = hipsolverDnDgeqrf(solverHandle, M, N, A_T->get_values(), lda, dTau, dWork, lwork, devInfo);
    cudaErrorCode = hipDeviceSynchronize();
    assert(cusolverStatus_SUCCESS == cusolverStatus);
    assert(hipSuccess == cudaErrorCode);

    // Copy A_T to R s.t. the upper triangle corresponds to R
    A_T->transpose(gko::lend(R));

    // Compute Q
    cusolverStatus = hipsolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, dWork, lwork, devInfo);
    cudaErrorCode = hipDeviceSynchronize();
    assert(cusolverStatus_SUCCESS == cusolverStatus);
    assert(hipSuccess == cudaErrorCode);

    A_T->transpose(gko::lend(Q));

    hipDeviceSynchronize();

    calculateQRDecompEvent.stop();

    temp_A->clear();

    return;
}
