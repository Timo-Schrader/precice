#include "hip/hip_runtime.h"
#include "ginkgo/ginkgo.hpp"
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "utils/Event.hpp"
#include "utils/EventUtils.hpp"

using GinkgoMatrix = gko::matrix::Dense<>;

// Handles for low-level CUDA libraries
hipsolverHandle_t solverHandle;
hipblasHandle_t cublasHandle;
hipblasStatus_t hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolverStatus = HIPSOLVER_STATUS_SUCCESS;
hipError_t cudaErrorCode = hipSuccess;

// Important variables which track the state of the solver routines
double *dTau = nullptr;
double *dWork = nullptr;
int *devInfo = nullptr;

void initCuSolver(){
    // Allocating important CUDA variables
    hipMalloc((void **)&dWork, sizeof(double));
    hipMalloc((void **)&devInfo, sizeof(int));
    hipMalloc((void **)&dTau, sizeof(double));

}

void deInitCuSolver(){
    // Freeing CUDA variables
    hipFree(dTau);
    hipFree(dWork);
    hipFree(devInfo);
}

void computeQR(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *A_Q, GinkgoMatrix *R)
{
    hipsolverDnCreate(&solverHandle);
    hipblasCreate(&cublasHandle);

    // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
    // Making a copy since every value will be overridden
    auto A_T = gko::share(GinkgoMatrix::create(exec, gko::dim<2>(A_Q->get_size()[1], A_Q->get_size()[0])));
    A_Q->transpose(gko::lend(A_T));

    // Setting dimensions for solver
    const unsigned int M = A_T->get_size()[1];
    const unsigned int N = A_T->get_size()[0];

    const int lda = max(1, M);
    const int k = min(M, N);

    int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;

    precice::utils::Event calculateQRDecompEvent{"calculateQRDecomp"};

    // Query working space of geqrf and orgqr
    cusolverStatus = hipsolverDnDgeqrf_bufferSize(solverHandle, M, N, A_T->get_values(), lda, &lwork_geqrf);
    assert(cusolverStatus == cusolverStatus_SUCCESS);
    cusolverStatus = hipsolverDnDorgqr_bufferSize(solverHandle, M, N, k, A_T->get_values(), lda, dTau, &lwork_orgqr);
    assert(cusolverStatus == cusolverStatus_SUCCESS);
    lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
    cudaErrorCode = hipMalloc((void **)&dWork, sizeof(double) * lwork);
    assert(hipSuccess == cudaErrorCode);

    // Compute QR factorization
    cusolverStatus = hipsolverDnDgeqrf(solverHandle, M, N, A_T->get_values(), lda, dTau, dWork, lwork, devInfo);
    cudaErrorCode = hipDeviceSynchronize();
    assert(cusolverStatus_SUCCESS == cusolverStatus);
    assert(hipSuccess == cudaErrorCode);

    // Copy A_T to R s.t. the upper triangle corresponds to R
    A_T->transpose(gko::lend(R));

    // Compute Q
    cusolverStatus = hipsolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, dWork, lwork, devInfo);
    cudaErrorCode = hipDeviceSynchronize();
    assert(cusolverStatus_SUCCESS == cusolverStatus);
    assert(hipSuccess == cudaErrorCode);

    A_T->transpose(gko::lend(A_Q));

    hipDeviceSynchronize();

    calculateQRDecompEvent.stop();

    return;
}
